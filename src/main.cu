#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <math.h>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

using namespace std;

__global__ void
countDis(float x, float y, float *x1, float *y1, float * result,  int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        result[i] = sqrt((x-x1[i])*(x-x1[i]) + (y-y1[i])*(y-y1[i]));

    }
}

float isKTheLowestNumInArr(thrust::device_vector<float> first, float kTheLowest,int k) {
	for(int i =0; i<k; i++){
		//cout << kTheLowest << " " <<first[i]<<endl;
		if (kTheLowest == first[i])
		   return 1;
		if(kTheLowest > first[i])
			return 0;
	}
	return 0;
}


int main(void) {
	const int nFPoints = 1000000;
	const int size = nFPoints;
	const float k = 5;

	float tDivider = (2000*(k-1));
	cout << (tDivider) << endl;
	float mean = 0.49974;
	float var = sqrt(0.0006);

	thrust::host_vector<float> x1(nFPoints);
	thrust::host_vector<float> y1(nFPoints);
	thrust::host_vector<float> x2(nFPoints);
	thrust::host_vector<float> y2(nFPoints);

	thrust::host_vector<float> hdisIn(1000);
	thrust::host_vector<float> hdisOut(1000);
	thrust::host_vector<float> ht(nFPoints);

	string line;

	std::fstream myfile("Gauss_1111.txt", std::ios_base::in);

	float a;

	for(int i =0; i < size; i++) {
		myfile >> a;
		x1[i] = a;
		myfile >> a;
		y1[i] = a;
		myfile >> a;
		x2[i] = a;
		myfile >> a;
		y2[i] = a;
	}

	thrust::device_vector<float> x1d = x1;
	thrust::device_vector<float> y1d = y1;
	thrust::device_vector<float> x2d = x2;
	thrust::device_vector<float> y2d = y2;

	thrust::device_vector<float> disIn = hdisIn;
	thrust::device_vector<float> disOut = hdisOut;
	thrust::device_vector<float> htd = ht;

	int threadsPerBlock = 257;
	int blocksPerGrid = (1000 + threadsPerBlock - 1) / threadsPerBlock;

	float t = 0;

	 for(int j = 0; j< 1000; j ++){
	 for(int i=0; i < 1000; i++) {
	    countDis<<<blocksPerGrid, threadsPerBlock>>>(x1[i+(j*1000)],y1[i+(j*1000)], thrust::raw_pointer_cast(&x1d[j*1000]), thrust::raw_pointer_cast(&y1d[j*1000]), thrust::raw_pointer_cast(&disIn[0]),1000);
	    countDis<<<blocksPerGrid, threadsPerBlock>>>(x1[i+(j*1000)],y1[i+(j*1000)], thrust::raw_pointer_cast(&x2d[j*1000]), thrust::raw_pointer_cast(&y2d[j*1000]), thrust::raw_pointer_cast(&disOut[0]),1000);
	    thrust::device_vector<float> disBoth = disIn;
	    disBoth.insert(disBoth.end(), disOut.begin(), disOut.end());

	    thrust::sort(disBoth.begin(),disBoth.end());
	    thrust::sort(disOut.begin(), disOut.end());
	    // thrust::host_vector<float> both = disBoth;
	    // thrust::host_vector<float> in = disIn;

	    for(int i = 0; i < k; i++) {
	    	 t += isKTheLowestNumInArr(disOut, disBoth[i],k);
	    }
	   // cout << i << endl;
	 }



	 for(int i=0; i < 1000; i++){
		 countDis<<<blocksPerGrid, threadsPerBlock>>>(x2[i+(j*1000)],y2[i+(j*1000)], thrust::raw_pointer_cast(&x1d[j*1000]), thrust::raw_pointer_cast(&y1d[j*1000]), thrust::raw_pointer_cast(&disOut[0]),1000);
		 countDis<<<blocksPerGrid, threadsPerBlock>>>(x2[i+(j*1000)],y2[i+(j*1000)], thrust::raw_pointer_cast(&x2d[j*1000]), thrust::raw_pointer_cast(&y2d[j*1000]), thrust::raw_pointer_cast(&disIn[0]),1000);
		 thrust::device_vector<float> disBoth = disIn;
		 disBoth.insert(disBoth.end(), disOut.begin(), disOut.end());

		 thrust::sort(disBoth.begin(),disBoth.end());
		 thrust::sort(disOut.begin(), disOut.end());
		 	// thrust::host_vector<float> both = disBoth;
		 	// thrust::host_vector<float> in = disIn;

		 for(int i = 1; i < k; i++) {
		 	t += isKTheLowestNumInArr(disOut, disBoth[i],k);
		 }
		 //cout << i << endl;
	 }


	 cout << ((t/tDivider) -mean)*var << endl;
	// cout << t << endl;
	 t = 0;
   }
	cout << "Koniec.";
	hipDeviceReset();
	return 0;
}
