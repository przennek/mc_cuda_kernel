#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <math.h>

using namespace std;

__global__ void
velStepRand(float *v, const float *a, float *rand, int numElements)
{
	 int i = blockDim.x * blockIdx.x + threadIdx.x;
	 if(i < numElements){

		 if (rand[i]<0.5)
		 {
			 v[i] = v[i] + a[i];

		 }
		 else{
			 v[i] = -v[i] + a[i];

		 }

	 }

}




int main(void) {
	cout << "aa";
	return 0;
}

