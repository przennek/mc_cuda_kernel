#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <math.h>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

using namespace std;

__global__ void
countDis(float x, float y, float *x1, float *y1, float * result,  int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        result[i] = sqrt((x-x1[i])*(x-x1[i]) + (y-y1[i])*(y-y1[i]));

    }
}


int main(void) {
	const int nFPoints = 1000000;
	const int size = nFPoints;

	thrust::host_vector<float> x1(nFPoints);
	thrust::host_vector<float> y1(nFPoints);
	thrust::host_vector<float> x2(nFPoints);
	thrust::host_vector<float> y2(nFPoints);

	thrust::host_vector<float> hdisIn(1000);
	thrust::host_vector<float> hdisOut(1000);
	thrust::host_vector<float> ht(nFPoints);

	string line;

	std::fstream myfile("Gauss_1111.txt", std::ios_base::in);

	float a;

	for(int i =0; i < size; i++) {
		myfile >> a;
		x1[i] = a;
		myfile >> a;
		y1[i] = a;
		myfile >> a;
		x2[i] = a;
		myfile >> a;
		y2[i] = a;
	}

	thrust::device_vector<float> x1d = x1;
	thrust::device_vector<float> y1d = y1;
	thrust::device_vector<float> x2d = x2;
	thrust::device_vector<float> y2d = y2;

	thrust::device_vector<float> disIn = hdisIn;
	thrust::device_vector<float> disOut = hdisOut;
	thrust::device_vector<float> t = ht;

	int threadsPerBlock = 257;
	int blocksPerGrid = (1000 + threadsPerBlock - 1) / threadsPerBlock;

	 //liczymy pojedyncze T
	 for(int i=0; i < 1000; i++) {
	    countDis<<<blocksPerGrid, threadsPerBlock>>>(x1[i],y1[i], thrust::raw_pointer_cast(&x1d[0]), thrust::raw_pointer_cast(&y1d[0]), thrust::raw_pointer_cast(&disIn[0]),1000);
	    countDis<<<blocksPerGrid, threadsPerBlock>>>(x1[i],y1[i], thrust::raw_pointer_cast(&x2d[0]), thrust::raw_pointer_cast(&y2d[0]), thrust::raw_pointer_cast(&disOut[0]),1000);
	    //do tablicy t na pozycje i wpisujemy ilosc sposrod k najblizszych dystatnsow z
	    //disIn, disOut, ktore znajduja sie w disOut
	    //countSingleT<<<blocksPerGrid, threadsPerBlock>>>(disIn, disOut, t , 1000, i,k);
	 }

	 for(int i=0; i < 1000; i++){
		 countDis<<<blocksPerGrid, threadsPerBlock>>>(x2[i],y2[i], thrust::raw_pointer_cast(&x1d[0]), thrust::raw_pointer_cast(&y1d[0]), thrust::raw_pointer_cast(&disIn[0]),1000);
		 countDis<<<blocksPerGrid, threadsPerBlock>>>(x2[i],y2[i], thrust::raw_pointer_cast(&x2d[0]), thrust::raw_pointer_cast(&y2d[0]), thrust::raw_pointer_cast(&disOut[0]),1000);
	    //do tablicy t na pozycje i + 1000 wpisujemy ilosc sposord k najblizszych dystatnsow z
	    //disIn, disOut, ktore znajduja sie w disOut
	    //countSingleT<<<blocksPerGrid, threadsPerBlock>>>(disIn, disOut, t , 1000, i +1000,k);
	 }
	 //suma tablicy t to pojedyncze T
	 thrust::sort(disOut.begin(),disOut.end());
	 thrust::host_vector<float> D = disOut;


	 for(int i = 0; i < 1000; i++) {
		 cout << D[i] << " ";
	 }
	 cout << "\n";

//	cout << "Koniec.";
	return 0;
}
