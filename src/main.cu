#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <math.h>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

using namespace std;

__global__ void
countDis(float x, float y, float *x1, float *y1, float * result,  int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        result[i] = sqrt((x-x1[i])*(x-x1[i]) + (y-y1[i])*(y-y1[i]));

    }
}

int isKTheLowestNumInArr(thrust::host_vector<float> first, float kTheLowest) {
	if(std::find(first.begin(), first.end(), x) != first.end()) {
	    return 1;
	} else {
	    return 0;
	}
}


int main(void) {
	const int nFPoints = 1000000;
	const int size = nFPoints;
	const int k = 5;

	thrust::host_vector<float> x1(nFPoints);
	thrust::host_vector<float> y1(nFPoints);
	thrust::host_vector<float> x2(nFPoints);
	thrust::host_vector<float> y2(nFPoints);

	thrust::host_vector<float> hdisIn(1000);
	thrust::host_vector<float> hdisOut(1000);
	thrust::host_vector<float> ht(nFPoints);

	string line;

	std::fstream myfile("Gauss_1111.txt", std::ios_base::in);

	float a;

	for(int i =0; i < size; i++) {
		myfile >> a;
		x1[i] = a;
		myfile >> a;
		y1[i] = a;
		myfile >> a;
		x2[i] = a;
		myfile >> a;
		y2[i] = a;
	}

	thrust::device_vector<float> x1d = x1;
	thrust::device_vector<float> y1d = y1;
	thrust::device_vector<float> x2d = x2;
	thrust::device_vector<float> y2d = y2;

	thrust::device_vector<float> disIn = hdisIn;
	thrust::device_vector<float> disOut = hdisOut;
	thrust::device_vector<float> t = ht;

	int threadsPerBlock = 257;
	int blocksPerGrid = (1000 + threadsPerBlock - 1) / threadsPerBlock;

	float t = 0;

	 //liczymy pojedyncze T
	 for(int i=0; i < 1000; i++) {
	    countDis<<<blocksPerGrid, threadsPerBlock>>>(x1[i],y1[i], thrust::raw_pointer_cast(&x1d[0]), thrust::raw_pointer_cast(&y1d[0]), thrust::raw_pointer_cast(&disIn[0]),1000);
	    countDis<<<blocksPerGrid, threadsPerBlock>>>(x1[i],y1[i], thrust::raw_pointer_cast(&x2d[0]), thrust::raw_pointer_cast(&y2d[0]), thrust::raw_pointer_cast(&disOut[0]),1000);
	    //do tablicy t na pozycje i wpisujemy ilosc sposrod k najblizszych dystatnsow z
	    //disIn, disOut, ktore znajduja sie w disOut
	    //countSingleT<<<blocksPerGrid, threadsPerBlock>>>(disIn, disOut, t , 1000, i,k);
	 }

	 thrust::device_vector<float> disBoth = disIn;
	 disBoth.insert(disBoth.end(), disOut.begin(), disOut.end());

	 thrust::sort(disBoth.begin(),disBoth.end());
	 thrust::host_vector<float> both = disBoth;
	 thrust::host_vector<float> in = disIn;

	 for(int i = 0; i < k; i++) {
		 t += isKTheLowestNumInArr(in, both[i]);
	 }

	 for(int i=0; i < 1000; i++){
		 countDis<<<blocksPerGrid, threadsPerBlock>>>(x2[i],y2[i], thrust::raw_pointer_cast(&x1d[0]), thrust::raw_pointer_cast(&y1d[0]), thrust::raw_pointer_cast(&disIn[0]),1000);
		 countDis<<<blocksPerGrid, threadsPerBlock>>>(x2[i],y2[i], thrust::raw_pointer_cast(&x2d[0]), thrust::raw_pointer_cast(&y2d[0]), thrust::raw_pointer_cast(&disOut[0]),1000);
	    //do tablicy t na pozycje i + 1000 wpisujemy ilosc sposord k najblizszych dystatnsow z
	    //disIn, disOut, ktore znajduja sie w disOut
	    //countSingleT<<<blocksPerGrid, threadsPerBlock>>>(disIn, disOut, t , 1000, i +1000,k);
	 }
	 //suma tablicy t to pojedyncze T



//	cout << "Koniec.";
	return 0;
}
