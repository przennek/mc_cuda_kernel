#include <iostream>
#include <fstream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <math.h>
#include <vector>

using namespace std;

const int LINE_LEN = 4003;

string* readLine(string line, int lineLen) {
	 string* tokensArr = new string[lineLen];
	 std::vector<std::string> tokens;
	 split(tokens, str, is_any_of(" "));
	 int counter = 0;
	 for(auto& s: tokens) {
		 tokensArr[counter++] = s;
	 }
	 return tokensArr;
}

void appendVec(string* line, int lineLen, vector<float>* data1X, vector<float>* data1Y, vector<float>* data2X, vector<float>* data2Y) {

}

int main(void) {
	const int nFPoints = 1000;
	const int rows = 4;
	float **data = (float**)malloc(rows * sizeof(float*));
	for(int i = 0; i < rows; i++) {
		data[i] = (float*)malloc(nFPoints * sizeof(float));
	}

	string line;
	ifstream myfile("Gauss_1111.txt");
	if (myfile.is_open()) {
		while (getline (myfile, line)) {
//			cout << line << '\n';
	    }
	    myfile.close();
	}
	else cout << "Unable to open file";

	for(int i = 0; i < rows; i++) {
		free(data[i]);
	}
	free(data);

	cout << "Koniec.";
	return 0;
}
