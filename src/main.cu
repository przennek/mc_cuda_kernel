#include <iostream>
#include <fstream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <math.h>
#include <vector>

using namespace std;

const int LINE_LEN = 4003;


__global__ void
countDis(float x, float y, float *x1, float *y1, float * result,  int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        result[i] = sqrt((x-x1[i])*(x-x1[i]) + (y-y1[i])*(y-y1[i]));

    }
}

__device__ void
countAllDistArr(float *x1, float *y1,float *x2, float *y2, float * result, int size){

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	 if (i < size*size)
	    {
	       // countDis(x1[i], y1[i], x2,y2, result + i*size,size);
	    }

}



int main(void) {
	const int nFPoints = 1000000;
	const int size = nFPoints;
	float * x1 = (float*)malloc(nFPoints * sizeof(float));
	float * y1 = (float*)malloc(nFPoints * sizeof(float));
	float * x2 = (float*)malloc(nFPoints * sizeof(float));
	float * y2 = (float*)malloc(nFPoints * sizeof(float));

	string line;

	std::fstream myfile("/home/gr/Pulpit/multicorev2/mc_fit/src/Gauss_1111.txt", std::ios_base::in);

	    float a;

	 for(int i =0; i < size; i++){
	    	myfile >> a;
	    	x1[i] = a;
	    	myfile >> a;
	        y1[i] = a;
	        myfile >> a;
	        x2[i] = a;
	    	myfile >> a;
	        y2[i] = a;
	 }



	 float *x1d = NULL;
	 hipMalloc((void **)&x1d, size);

	 float *y1d = NULL;
	 hipMalloc((void **)&y1d, size);

	 float *x2d = NULL;
	 hipMalloc((void **)&x2d, size);

	 float *y2d = NULL;
	 hipMalloc((void **)&y2d, size);

	 float *disIn = NULL;
	 hipMalloc((void **)&disIn, 1000);
	 float *disOut = NULL;
	 hipMalloc((void **)&disOut, 1000);
	 float *t = NULL;
	 hipMalloc((void **)&t, 2000);


	 hipMemcpy(x1d, x1, size, hipMemcpyHostToDevice);
	 hipMemcpy(y1d, y1, size, hipMemcpyHostToDevice);
	 hipMemcpy(x2d, x2, size, hipMemcpyHostToDevice);
	 hipMemcpy(y2d, y2, size, hipMemcpyHostToDevice);




	    int threadsPerBlock = 257;
	    int blocksPerGrid =(1000 + threadsPerBlock - 1) / threadsPerBlock;

	 //liczymy pojedyncze T
	    for(int i =0; i < 1000 ; i++){
	    	countDis<<<blocksPerGrid, threadsPerBlock>>>(x1[i],y1[i],x1d,y1d,disIn,1000);
	    	countDis<<<blocksPerGrid, threadsPerBlock>>>(x1[i],y1[i],x2d,y2d,disOut,1000);
	    	//do tablicy t na pozycje i wpisujemy ilosc sposrod k najblizszych dystatnsow z
	    	//disIn, disOut, ktore znajduja sie w disOut
	    	//countSingleT<<<blocksPerGrid, threadsPerBlock>>>(disIn, disOut, t , 1000, i,k);
	    }
	    for(int i =0; i < 1000 ; i++){
	    	countDis<<<blocksPerGrid, threadsPerBlock>>>(x2[i],y2[i],x1d,y1d,disOut,1000);
	        countDis<<<blocksPerGrid, threadsPerBlock>>>(x2[i],y2[i],x2d,y2d,disIn,1000);
	        //do tablicy t na pozycje i + 1000 wpisujemy ilosc sposord k najblizszych dystatnsow z
	        //disIn, disOut, ktore znajduja sie w disOut
	        //countSingleT<<<blocksPerGrid, threadsPerBlock>>>(disIn, disOut, t , 1000, i +1000,k);
	    }
	    //suma tablicy t to pojedyncze T




	free(x1);
	free(y1);
	free(x2);
	free(y2);

	cout << "Koniec.";
	return 0;
}
